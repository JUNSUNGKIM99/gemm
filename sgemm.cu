#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "cublas_utils.h"

using data_type = double;

void randomInit(float *data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;

}

int main(int argc, char *argv[])
{
    if( argc != 5){
        printf("Usage: ./cublas_sgemm m n k iter \n");
        printf("Current # of arguments: %d\n", argc);
        return 0;
    }
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    const int m = atoi(argv[1]);
    const int n = atoi(argv[2]);
    const int k = atoi(argv[3]);
    const int nIter = atoi(argv[4]);
    printf("M: %d, N: %d, K: %d\n", m, n, k);
    const int lda = m;
    const int ldb = k;
    const int ldc = m;


    double size_A = m * k;
    double mem_size_A = sizeof(float) * size_A;
    float *h_A = (float *)malloc(mem_size_A);
    double size_B = n * k;
    double mem_size_B = sizeof(float) * size_B;
    float *h_B = (float *)malloc(mem_size_B);

    randomInit(h_A, size_A);
    randomInit(h_B, size_B);

    float *d_A, *d_B, *d_C;

    unsigned int size_C = m * n;
    unsigned int mem_size_C = sizeof(float) * size_C;

    float *h_C = (float *)malloc(mem_size_C);
    float *h_CUBLAS = (float *)malloc(mem_size_C);

    CUDA_CHECK(hipMalloc((void **)&d_A, mem_size_A));
    CUDA_CHECK(hipMalloc((void **)&d_B, mem_size_B));
    CUDA_CHECK(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc((void **)&d_C, mem_size_C));
    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */

    CUDA_CHECK(hipMalloc((void **)&d_A, mem_size_A));
    CUDA_CHECK(hipMalloc((void **)&d_B, mem_size_B));
    CUDA_CHECK(hipMemcpyAsync(d_A, h_A, mem_size_A, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMalloc((void **)&d_C, mem_size_C));

    /*Setup to Execution*/
    //    hipblasHandle_t handle;
    hipEvent_t start, stop;
    //    CUDA_CHECK(hipblasCreate(&handle));
    /* step 3: compute */
    // Warm up kernel execution
    CUBLAS_CHECK(
        hipblasSgemm(cublasH, transa, transb, m, n, k, &alpha, d_A, lda, d_B, ldb, &beta, d_C, ldc));

    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start, NULL));

    for (int j = 0; j < nIter; j++)
    {
        CUBLAS_CHECK(
            hipblasSgemm(cublasH, transa, transb, m, n, k, &alpha, d_A, lda, d_B, ldb, &beta, d_C, ldc));
    }
    printf("Done.\n");

    CUDA_CHECK(hipEventRecord(stop, NULL));
    CUDA_CHECK(hipEventSynchronize(stop));
    float msecTotal = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&msecTotal, start, stop));

    float msecPerMatrixMul = msecTotal / nIter;
    double flopsPerMatrixMul = 2.0 * (double)m * (double)n * (double)k;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);

    printf("MatrixMul size, M=%d N=%d K=%d\n", m, n, k);
    printf(
        "Performance= %.4f GFlops/s, Time= %.4f msec, Size= %.0f Ops\n",
        gigaFlops, msecPerMatrixMul, flopsPerMatrixMul);

    /* step 4: copy data to host */
    CUDA_CHECK(hipMemcpyAsync(h_CUBLAS, d_C, mem_size_C, hipMemcpyDeviceToHost, stream));\
    CUDA_CHECK(hipStreamSynchronize(stream));

    /* free resources */
    free(h_A);
    free(h_B);
    free(h_C);
    
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    
    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
